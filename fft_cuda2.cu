#include "hip/hip_runtime.h"
#include <fft_cuda.cuh>

__global__ void fft_cuda2_kernel(complex_t *ip, complex_t *op, int m, int size)
{
  __shared__ complex_t shared_op[2048];
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  if (tid == 0) {
    for (int i = 0, j = 0, k; i < size-1; i++) {
      if (i <= j) {
        shared_op[j] = ip[i];
        shared_op[i] = ip[j];
      }

      for (k = size/2; k <= j; k >>= 1) {
        j -= k;
      }
      j += k;
    }

    shared_op[size-1] = ip[size-1];
  }

  __syncthreads();

  for (int i = 0; i < m; i++) {
    int len = 1 << i;  /* the length of half block at level m*/
    complex_t factor = {cos(-2.0 * PI / (2 * len)), sin(-2.0 * PI / (2 * len))};

    int block_len = (len << 1);
    int nblocks = size / block_len;

    if (tid < nblocks) {
      int j = tid * block_len;
      complex_t omega = {1, 0};

      for (int k = j; k < j+len; k++) {
        complex_t temp = cuda_complex_mult(omega, shared_op[k+len]);

        shared_op[k+len] = cuda_complex_sub(shared_op[k], temp);
        shared_op[k    ] = cuda_complex_add(shared_op[k], temp);

        omega = cuda_complex_mult(omega, factor);
      }
    }

    __syncthreads();
  }

  for (; tid < size; tid += blockDim.x) {
    op[tid] = shared_op[tid];
  }
}

void fft_cuda2(complex_t *_ip, complex_t *_op, int size)
{
  int m = (int)log2((double)size);
  complex_t *ip = (complex_t *)_ip;
  complex_t *op = (complex_t *)_op;

  gpuErrchk(hipMemcpy(dev_ip, ip, size*sizeof(complex_t), hipMemcpyHostToDevice));
 
  /* Can only work until size 2048 */
  int threads = (1024 < size) ? 1024 : size;
  dim3 block(threads, 1, 1);
  dim3 grid(size/threads, 1, 1);

  fft_cuda2_kernel<<<grid, block>>> (dev_ip, dev_op, m, size);
  gpuErrchk(hipPeekAtLastError());

  gpuErrchk(hipMemcpy(op, dev_op, size*sizeof(complex_t), hipMemcpyDeviceToHost));
}
